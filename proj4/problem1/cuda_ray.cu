#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define CUDA 0
#define OPENMP 1
#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048



typedef struct Sphere {
    float r, g, b;
    float radius;
    float x, y, z;
} Sphere;

__device__ float hit(Sphere* sphere, float ox, float oy, float *n)
{
    float dx = ox - sphere->x;
    float dy = oy - sphere->y;
    if (dx * dx + dy * dy < sphere->radius * sphere->radius) {
        float dz = sqrtf(sphere->radius * sphere->radius - dx * dx - dy * dy);
        *n = dz / sqrtf(sphere->radius * sphere->radius);
        return dz + sphere->z;
    } else 
        return -INF;
}

__global__ void kernel(int x, int y, Sphere* s, unsigned char* ptr)
{
    int offset = x + y*DIM;
    float ox = (x - DIM/2);
    float oy = (y - DIM/2);

    // printf("x:%d, y:%d, ox:%f, oy:%f\n",x,y,ox,oy);

    float r=0, g=0, b=0;
    float   maxz = -INF;
    for(int i=0; i<SPHERES; i++) {
        float n;
        float t = hit(&s[i], ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset*4 + 0] = (int)(r * 255);
    ptr[offset*4 + 1] = (int)(g * 255);
    ptr[offset*4 + 2] = (int)(b * 255);
    ptr[offset*4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim,int ydim, FILE* fp)
{
    int i,x,y;
    fprintf(fp,"P3\n");
    fprintf(fp,"%d %d\n",xdim, ydim);
    fprintf(fp,"255\n");
    for (y=0;y<ydim;y++) {
        for (x=0;x<xdim;x++) {
            i=x+y*xdim;
            fprintf(fp,"%d %d %d ",bitmap[4*i],bitmap[4*i+1],bitmap[4*i+2]);
        }
        fprintf(fp,"\n");
    }
}

int main(int argc, char* argv[])
{
    int x,y;
    unsigned char* bitmap;

    srand(time(NULL));

    if (argc != 1) {
        printf("Too many arguments\n");
        exit(0);
    }
    FILE* fp = fopen("result.ppm","w");
    
    Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
    for (int i=0; i<SPHERES; i++) {
        temp_s[i].r = rnd( 1.0f );
        temp_s[i].g = rnd( 1.0f );
        temp_s[i].b = rnd( 1.0f );
        temp_s[i].x = rnd( 2000.0f ) - 1000;
        temp_s[i].y = rnd( 2000.0f ) - 1000;
        temp_s[i].z = rnd( 2000.0f ) - 1000;
        temp_s[i].radius = rnd( 200.0f ) + 40;
    }
    
    bitmap=(unsigned char*)malloc(sizeof(unsigned char)*DIM*DIM*4);
    for (x=0;x<DIM;x++)
        for (y=0;y<DIM;y++)
            kernel<<<1, 1>>>(x,y,temp_s,bitmap);
    ppm_write(bitmap,DIM,DIM,fp);
    // printf("Openmp (%i threads): %fms\n", no_threads, total_time * 1000);
    // printf("[%s.ppm] generated\n", argv[2]);

    fclose(fp);
    free(bitmap);
    free(temp_s);

    return 0;
}